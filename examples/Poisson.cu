#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2019-2020. Triply periodic Poisson example.
Computes the electric field between two opposite charges placed in a periodic box.
USAGE:
./poisson [L] [r] [gw]

gw: Gaussian width of the charges
L: Box size (cubic box)
r: distance between the charges

In the limit when L->inf the field between the two particles should be:
Ex =  exp(-r**2/(4.0*gw**2))/(4*pi**1.5*gw*r) - erf(r/(2.0*gw))/(4*pi*r**2);

*/

//This include contains the basic needs for an uammd project
#include"uammd.cuh"
#include"Interactor/SpectralEwaldPoisson.cuh"
#include<fstream>

using namespace uammd;
using std::make_shared;
using std::endl;

int main(int argc, char *argv[]){
  int N = 2;
  real gw = std::stod(argv[3]);
  real L = std::stod(argv[1]);
  real r = std::stod(argv[2]);
  auto sys = make_shared<System>(argc, argv);
  auto pd = make_shared<ParticleData>(N, sys);
  Box box(L);
  {
    auto pos = pd->getPos(access::location::cpu, access::mode::write);
    auto charge = pd->getCharge(access::location::cpu, access::mode::write);
    pos[0] = make_real4(-r*0.5,0,0,0);
    pos[1] = make_real4( r*0.5,0,0,0);
    charge[0] = 1;
    charge[1] = -1;
  }
  auto pg = make_shared<ParticleGroup>(pd, sys, "All");
  Poisson::Parameters par;
  par.box = box;
  par.epsilon = 1;
  par.gw = gw;
  par.tolerance = 1e-8;
  par.split = std::stod(argv[4]);
  auto poisson = make_shared<Poisson>(pd, pg, sys, par);
  {
    auto force = pd->getForce(access::location::gpu, access::mode::write);
    thrust::fill(thrust::cuda::par, force.begin(), force.end(), real4());
    auto energy = pd->getEnergy(access::location::gpu, access::mode::write);
    thrust::fill(thrust::cuda::par, energy.begin(), energy.end(), real());
  }
  poisson->sumForce(0);
  //poisson->sumEnergy();
  {
    auto pos = pd->getPos(access::location::cpu, access::mode::read);
    auto energy = pd->getEnergy(access::location::cpu, access::mode::read);
    auto force = pd->getForce(access::location::cpu, access::mode::read);
    auto charge = pd->getCharge(access::location::cpu, access::mode::read);
    real3 p;
    fori(0,N){
      real4 pc = pos[i];
      p = make_real3(pc);
      int type = charge[i];
      std::cout<<std::setprecision(15)<<p<<" q: "<<charge[i]<<" F: "<<force[i]<<endl;
    }
  }
  sys->finish();
  return 0;
}


