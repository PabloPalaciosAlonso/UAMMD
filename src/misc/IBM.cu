#include "hip/hip_runtime.h"
#include"IBM.cuh"
#include<third_party/type_names.h>

namespace uammd{

  namespace IBM_ns{

#ifndef SINGLE_PRECISION
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ < 600 
    __device__ double atomicAdd(double* address, double val){
      unsigned long long int* address_as_ull =
	(unsigned long long int*)address;
      unsigned long long int old = *address_as_ull, assumed;
      do {
	assumed = old;
	old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
					     __longlong_as_double(assumed)));
      } while (assumed != old);
      return __longlong_as_double(old);
    }
#endif
#endif


    inline __device__ real3 atomicAdd(real3* address, real3 val){
      real3 newval;
      if(val.x) newval.x = atomicAdd(&(*address).x, val.x);
      if(val.y) newval.y = atomicAdd(&(*address).y, val.y);
      if(val.z) newval.z = atomicAdd(&(*address).z, val.z);
      return newval;
    }

    inline __device__ real2 atomicAdd(real2* address, real2 val){
      real2 newval;
      if(val.x) newval.x = atomicAdd(&(*address).x, val.x);
      if(val.y) newval.y = atomicAdd(&(*address).y, val.y);
      return newval;
    }

    /*Spreads the 3D quantity v (defined on the particle positions) to a grid
      
      S v(z) = v(x) = \sum_{z}{ v(z)*\delta(||z-x||^2) }
      Where:
      - S is the spreading operator
      - "v" is a certain quantity
      - "z" is the position of the particles
      - "x" is the position of a grid cell
      - \delta() is the window function
    */
    template<bool is2D, class Grid, class Kernel,
      class PosIterator,
      class ParticleQuantityIterator, class GridQuantityIterator>
    __global__ void particles2GridD(const PosIterator __restrict__ pos, /*Particle positions*/
				    const ParticleQuantityIterator __restrict__ v,   /*Per particle quantity to spread*/
				    GridQuantityIterator  __restrict__ gridQuantity, /*Spreaded values, size ncells*/
				    int N, /*Number of particles*/
				    Grid grid, /*Grid information and methods*/
				    Kernel kernel){
      const int id = blockIdx.x;
      const int tid = threadIdx.x;
      using QuantityType = typename std::iterator_traits<GridQuantityIterator>::value_type;
      using ParticleQuantityType = typename std::iterator_traits<ParticleQuantityIterator>::value_type;
      
      if(id>=N) return;

      __shared__ real3 pi;
      __shared__ ParticleQuantityType vi; //The quantity for particle id
      __shared__ int3 celli;
      __shared__ int3 P; //Neighbour cell offset
      if(tid==0){
	pi = make_real3(pos[id]);
	vi = v[id];
	celli = grid.getCell(pi);
	const auto invCellSize = real(1.0)/grid.getCellSize(celli);
	  
	P = make_int3(kernel.support/2);
	//Kernels with even support might need an offset of one cell depending on the position of the particle inside the cell
	if(kernel.support%2==0){
	  const real3 pi_pbc = grid.box.apply_pbc(pi);
	  P -= make_int3( (pi_pbc+grid.box.boxSize*real(0.5))*invCellSize - make_real3(celli) + real(0.5) );
	}
	if(is2D) P.z = 0;
      }
      const int supportCells = kernel.support;
      int numberNeighbourCells = supportCells*supportCells;
      if(!is2D)  numberNeighbourCells *= supportCells;
      __syncthreads();
      for(int i = tid; i<numberNeighbourCells; i+=blockDim.x){
	/*Compute neighbouring cell*/
	int3 cellj = make_int3(celli.x + i%supportCells - P.x,
			       celli.y + (i/supportCells)%supportCells - P.y,
			       is2D?0:(celli.z + i/(supportCells*supportCells) - P.z));
	cellj = grid.pbc_cell(cellj);
	  
	/*Distance from particle i to center of cell j*/
	const real3 rij = grid.distanceToCellCenter(pi, cellj);
	const real k = kernel.delta(rij, grid.getCellSize(cellj));
	/*The weight of particle i on cell j*/
	const auto weight = vi*kernel.delta(rij, grid.getCellSize(cellj));
	// if(weight.x) printf("celli: %d %d , cellj: %d %d , weight: %g %g\n",
	// 		    celli.x, celli.y,
	// 		    cellj.x, cellj.y,
	// 		    weight.x, weight.y);
	/*Get index of cell j*/
	const int jcell = grid.getCellIndex(cellj);
	  
	/*Atomically sum my contribution to cell j*/
	atomicAdd(&gridQuantity[jcell], weight);
      }
    }

    /*Interpolates a quantity (i.e velocity) from its values in the grid to the particles.

      J(z) q(x) = q(z) = \sum_{x\in G}{ q(x)*\delta(||x-z||^2) weight(x)}
      Where :
         - J is the interpolation operator
	 - "q" a certain quantity
	 - "x" a cell of the grid
	 - "z" the position of a particle 
	 - \delta() is the window function given by Kernel
	 - weight() is the quadrature weight of a cell. (cellsize^d in a regular grid) given by QuadratureWeights

      This is the discretization of an integral and thus requires quadrature weigths for each element. 
        Which in a regular grid is just the cell size, h. But can in general be something depending on the position.
    */

    template<int TPP, bool is2D, class Grid,
      class Kernel,
      class PosIterator, class ResultIterator, class GridQuantityIterator,
      class QuadratureWeights>
    __global__ void grid2ParticlesDTPP(const PosIterator pos, /*Particle positions*/
				       ResultIterator Jq, /*Result for each particle*/
				       const GridQuantityIterator gridQuantity, /*Values in the grid*/
				       int N, /*Number of particles*/
				       Grid grid, /*Grid information and methods*/				  
				       Kernel kernel, 
				       QuadratureWeights qw /*Quadrature weights*/
				       ){
      const int id = blockIdx.x;
      const int tid = threadIdx.x;
      
      
      using GridQuantityType = typename std::iterator_traits<GridQuantityIterator>::value_type;
      using BlockReduce = hipcub::BlockReduce<GridQuantityType, TPP>;

      GridQuantityType result = GridQuantityType();
      
      __shared__ real3 pi;
      __shared__ int3 celli;
      __shared__ int3 P; //Neighbour cell offset
      __shared__ typename BlockReduce::TempStorage temp_storage;

      if(id<N){
	if(tid==0){
	  pi = make_real3(pos[id]);
	  celli = grid.getCell(pi);
	    
	  P = make_int3(kernel.support/2);
	  //Kernels with even support might need an offset of one cell depending on the position of the particle inside the cell
	  if(kernel.support%2==0){
	    const real3 invCellSize = real(1.0)/grid.getCellSize(celli);
	    const real3 pi_pbc = grid.box.apply_pbc(pi);
	    P -= make_int3( (pi_pbc+grid.box.boxSize*real(0.5))*invCellSize - make_real3(celli) + real(0.5) );
	  }
	  if(is2D) P.z = 0;
	}
      }
      __syncthreads();
      if(id<N){
	const int supportCells = kernel.support;
	int numberNeighbourCells = supportCells*supportCells;
	if(!is2D)  numberNeighbourCells *= supportCells;
	
	for(int i = tid; i<numberNeighbourCells; i+=blockDim.x){
	  //current neighbour cell
	  int3 cellj = make_int3(celli.x + i%supportCells - P.x,
				 celli.y + (i/supportCells)%supportCells - P.y,
				 is2D?0:(celli.z + i/(supportCells*supportCells) - P.z));
	  cellj = grid.pbc_cell(cellj);

	  //Compute distance to cell center
	  const real3 rij = grid.distanceToCellCenter(pi, cellj);
	  
	  const real weight = kernel.delta(rij, grid.getCellSize(cellj));

	  if(weight){
	    //J = S^T = St = σ S 
	    const int jcell = grid.getCellIndex(cellj);
	    const auto cellj_vel = gridQuantity[jcell];
	    const real dV = qw(cellj, grid);
	    // printf("i: %d ; cellj: %d %d %d ; celli: %d %d %d; rij: %g %g %g; weight: %.17g ; dV: %g\n",
	    //  	   id, cellj.x, cellj.y, cellj.z, celli.x, celli.y, celli.z, rij.x, rij.y, rij.z, weight, dV);
	    result += (dV*weight)*cellj_vel;
	  }
	}
      }
	  
      //Write total to global memory
      GridQuantityType total = BlockReduce(temp_storage).Sum(result);
      __syncthreads();
      if(tid==0 and id<N){
	using ResultType = typename std::iterator_traits<ResultIterator>::value_type;
	Jq[id] += static_cast<ResultType>(total);
      }
    }

  }
  
  template<class Kernel>
  IBM<Kernel>::IBM(shared_ptr<System> sys, shared_ptr<Kernel> kern):
    sys(sys), kernel(kern){
    sys->log<System::MESSAGE>("[IBM] Initialized with kernel: %s", type_name<Kernel>().c_str());
  }

  template<class Kernel>
  template<class Grid, class PosIterator, class QuantityIterator, class GridDataIterator>
  void IBM<Kernel>::spread(const PosIterator &pos, const QuantityIterator &v,
			   GridDataIterator &gridVels,
			   Grid grid, int numberParticles, hipStream_t st){
    sys->log<System::DEBUG2>("[IBM] Spreading");
    //Launch a small block per particle
    {
      int support = kernel->support;
      int numberNeighbourCells = support*support*support;
      int threadsPerParticle = std::min(32*(numberNeighbourCells/32), 512);
      if(numberNeighbourCells < 64) threadsPerParticle = 32;

      if(grid.cellDim.z == 1)
	IBM_ns::particles2GridD<true><<<numberParticles, threadsPerParticle, 0, st>>>
	  (pos, v, gridVels, numberParticles, grid, *kernel);
      else
	IBM_ns::particles2GridD<false><<<numberParticles, threadsPerParticle, 0, st>>>
	  (pos, v, gridVels, numberParticles, grid, *kernel);

    }
  }

  namespace IBM_ns{
    struct DefaultQuadratureWeights{
      inline __host__ __device__ real operator()(int3 cellj, const Grid &grid) const{
	return grid.getCellVolume(cellj);
      }
    };
  }
  template<class Kernel>
  template<class Grid,
      class PosIterator, class ResultIterator, class GridQuantityIterator>
  void IBM<Kernel>::gather(const PosIterator &pos, const ResultIterator &Jq,
			   const GridQuantityIterator &gridData,
			   Grid & grid, int numberParticles, hipStream_t st){
    IBM_ns::DefaultQuadratureWeights qw;
    this->gather(pos, Jq, gridData, grid, qw, numberParticles, st);
  }
  template<class Kernel>
  template<class Grid,
    class PosIterator, class ResultIterator, class GridQuantityIterator,
    class QuadratureWeights>
  void IBM<Kernel>::gather(const PosIterator &pos, const ResultIterator &Jq,
			   const GridQuantityIterator &gridData,
			   Grid & grid, const QuadratureWeights &qw, int numberParticles, hipStream_t st){
    if(grid.cellDim.z == 1)
      gather<true>(pos, Jq, gridData, grid, qw, numberParticles, st);    
    else
      gather<false>(pos, Jq, gridData, grid, qw, numberParticles, st);
    

  }

  template<class Kernel>
  template<bool is2D, class Grid,
    class PosIterator, class ResultIterator, class GridQuantityIterator,
    class QuadratureWeights>
  void IBM<Kernel>::gather(const PosIterator &pos, const ResultIterator &Jq,
			   const GridQuantityIterator &gridData,
			   Grid & grid, const QuadratureWeights &qw, int numberParticles, hipStream_t st){
    sys->log<System::DEBUG2>("[IBM] Gathering");
    
    int support = kernel->support;
    int numberNeighbourCells = support*support*support;
    int threadsPerParticle = std::min(int(pow(2,int(std::log2(numberNeighbourCells)+0.5))), 512);
    if(numberNeighbourCells < 64) threadsPerParticle = 32;

    auto grid2Particles = IBM_ns::grid2ParticlesDTPP<32, is2D, Grid, Kernel, PosIterator,  ResultIterator, GridQuantityIterator, QuadratureWeights>;

#define KERNEL(x) else if(threadsPerParticle<=x) grid2Particles = IBM_ns::grid2ParticlesDTPP<x, is2D, Grid, Kernel, PosIterator, ResultIterator, GridQuantityIterator, QuadratureWeights>;
    if(threadsPerParticle<=32){}
      KERNEL(64)
      KERNEL(128)
      KERNEL(256)
      KERNEL(512)
#undef KERNEL
	
      grid2Particles<<<numberParticles, threadsPerParticle, 0, st>>>(pos, Jq, gridData,
								     numberParticles, grid, *kernel, qw);



  }
  
}


