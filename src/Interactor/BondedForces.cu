#include "hip/hip_runtime.h"
/*
  Raul P. Pelaez 2016. Bonded pair forces Interactor implementation. i.e two body springs

  See BondedForces.cuh for more info.

TODO:
100 - Implement a warp reduction in computeForces
100- Implement sumEnergy and sumVirial
*/

#include"BondedForces.cuh"
#include<hipcub/hipcub.hpp>
#include<thrust/sort.h>
#include<thrust/reduce.h>
#include<algorithm>
#include<cstdint>
#include<vector>
#include<set>
#include<fstream>
namespace uammd{
  template<class BondType>
  BondedForces<BondType>::BondedForces(shared_ptr<ParticleData> pd,
				       shared_ptr<System> sys,
				       Parameters par,
				       BondType bondForce):
    Interactor(pd,
	       std::make_shared<ParticleGroup>(pd, sys, "All"),
	       sys,
	       "BondedForces/" + type_name<BondType>()),
    bondForce(bondForce), TPP(64){
    
    //BondedForces does not care about any parameter update, but the BondType might.
    this->setDelegate(&(this->bondForce));

    int numberParticles = pg->getNumberParticles();
    
    sys->log<System::MESSAGE>("[BondedForces] Initialized");

    sys->log<System::MESSAGE>("[BondedForces] Using: %s", type_name<BondType>().c_str());
    nbonds = nbondsFP = 0;
    /*If some bond type number is zero, the loop will simply not be entered, and no storage will be used*/
    /*Read the bond list from the file*/
    std::ifstream in(par.file);
    if(!in.good()){
      sys->log<System::CRITICAL>("[BondedForces] Bond file %s not found!!", par.file.c_str());
    }
    in>>nbonds;
    if(nbonds>0){
      bondList.resize(nbonds*2);//Allocate 2*nbonds, see init for explication
      thrust::host_vector<Bond> h_bondList = bondList;
      fori(0, nbonds){
	if(in.eof()){
	  sys->log<System::CRITICAL>("[BondedForces] ERROR! Bond file ended too soon! Expected %d lines, found %d", nbonds, i);
	}
	in>>h_bondList[i].i>>h_bondList[i].j;
	if(h_bondList[i].i >= numberParticles or h_bondList[i].j >= numberParticles)
	  sys->log<System::WARNING>("[BondedForces] Bond %d involves particles with index beyond the total number of particles!. i: %d, j:%d, N: %d", i, h_bondList[i].i, h_bondList[i].j, numberParticles);
	h_bondList[i].bond_info = BondType::readBond(in);
      }
      bondList = h_bondList;
    }
    sys->log<System::MESSAGE>("[BondedForces] %d particle-particle bonds detected.", bondList.size()/2);
    /*Fixed point bonds*/
    if(!in) nbondsFP = 0;
    else in>>nbondsFP;
    if(nbondsFP>0){
      bondListFP.resize(nbondsFP);
      thrust::host_vector<BondFP> h_bondListFP = bondListFP;
      fori(0, nbondsFP){
	in>>h_bondListFP[i].i;
	if(h_bondListFP[i].i >= numberParticles)
	  sys->log<System::WARNING>("[BondedForces] Bond %d involves a particle with index beyond the total number of particles!. i: %d, N: %d", i, h_bondListFP[i].i, numberParticles);
	in>>h_bondListFP[i].pos.x>>h_bondListFP[i].pos.y>>h_bondListFP[i].pos.z;
	h_bondListFP[i].bond_info = BondType::readBond(in);
      }
      bondListFP = h_bondListFP;
    }

    sys->log<System::MESSAGE>("[BondedForces] Detected: %d particle-particle bonds and %d Fixed Point bonds",
			      bondList.size()/2, bondListFP.size());

    /*Upload and init GPU*/
    init();  
  }


  template<class BondType>
  BondedForces<BondType>::~BondedForces(){
    hipDeviceSynchronize();
    sys->log<System::MESSAGE>("[BondedForces] Destroyed");
  }

  namespace BondedForces_ns{
    //Criterion to sort bonds

    template<class BondType>
    struct BondComp{
      __device__ __host__ bool operator()(const BondType &a, const BondType &b){
	return a.i<b.i;
      }
    };

    template<class Bond>
    //Takes a bondList that is filled from 0 to nbonds and mirrors it in nbonds 2nbonds
    __global__ void dupicateBonds(Bond * bondList,
				  int nbonds){
      int id = blockIdx.x*blockDim.x + threadIdx.x;
      if(id >= nbonds) return;
      int i = id + nbonds;
      Bond b = bondList[i-nbonds];
      thrust::swap(b.i, b.j);
      bondList[i] = b;
    }


  }


  //Initialize variables and upload them to GPU, init CUDA
  template<class BondType>
  void BondedForces<BondType>::init(){
    if(nbonds > 0)
      this->initParticleParticle();
    if(nbondsFP > 0)
      this->initFixedPoint();
    sys->log<System::MESSAGE>("[BondedForces] %d particles have at least one bond",
			      std::max(bondStart.size(), bondListFP.size()));
  }

  template<class BondType>
  void BondedForces<BondType>::initParticleParticle(){
    int numberParticles = pg->getNumberParticles();
    // ****************************************Pair bonds********************************************* 
    //This algorithm is identical to the one used in PairForces to sort by cell
    //First store all bonded pairs. That means i j and j i
    //The first ones are readed given, the complementary have to be generated
    
    int BLOCKSIZE = 128;
    int Nthreads = BLOCKSIZE<nbonds?BLOCKSIZE:nbonds;
    int Nblocks  =  nbonds/Nthreads +  ((nbonds%Nthreads!=0)?1:0); 

    sys->log<System::DEBUG>("[BondedForces] Duplicating bonds");
    auto d_bondList = thrust::raw_pointer_cast(bondList.data());
    
    BondedForces_ns::dupicateBonds<<<Nblocks, Nthreads>>>(d_bondList, nbonds);

    sys->log<System::DEBUG>("[BondedForces] Sorting bonds");
    /*Sort in the i index to construct bondStart and bondEnd*/
    thrust::sort(bondList.begin(), bondList.end(), BondedForces_ns::BondComp<Bond>());
    
    nbonds = bondList.size();
    //We have a list of bonds ordered by its first particle, so; All the particles
    // bonded with particle i=0, all particles "" i=1...

    //We need additional arrays to know where in the list the bonds of particle i start
    // and end
    //Initially all bondStarts are 2^32-1, this value means no particles bonded
    sys->log<System::DEBUG>("[BondedForces] Computing number of particles with bonds");
    thrust::host_vector<Bond> h_bondList = bondList;
    std::set<int> particlesWithBonds;
    fori(0, h_bondList.size()){
      particlesWithBonds.insert(h_bondList[i].i);
    }
    
    int nParticlesWithBonds = particlesWithBonds.size();
    sys->log<System::DEBUG>("[BondedForces] %d particles with bonds found", nParticlesWithBonds);
    
    bondStart.resize(nParticlesWithBonds, nullptr);
    nbondsPerParticle.resize(nParticlesWithBonds, 0);

    thrust::host_vector<Bond*> h_bondStart = bondStart;
    thrust::host_vector<int> h_nbondsPerParticle = nbondsPerParticle;
    sys->log<System::DEBUG>("[BondedForces] Filling bondStart");

    //Fill helper data structures
    int index = 0;
    int nbondsi = 0;
    for(int b = 0; b<nbonds; b++){
      int i = h_bondList[b].i;
      int inext;
      if(b<nbonds-1) inext = h_bondList[b+1].i;
      else inext = -1;
      nbondsi++;
      if(inext != i){
	if(index == 0)
	  h_bondStart[0] = thrust::raw_pointer_cast(bondList.data());    
	else
	  h_bondStart[index] = thrust::raw_pointer_cast(bondList.data())+b+1-nbondsi;
	
	h_nbondsPerParticle[index] = nbondsi;

	index++;
	nbondsi = 0;
      }
    }
    bondStart = h_bondStart;
    nbondsPerParticle = h_nbondsPerParticle;
    int meanBondsPerParticle = thrust::reduce(nbondsPerParticle.begin(), nbondsPerParticle.end())/bondStart.size();
    TPP = std::min((meanBondsPerParticle/32)*32, 128);
    TPP = std::max(TPP, 32);
    sys->log<System::MESSAGE>("[BondedForces] Mean bonds per particle: %d", meanBondsPerParticle);
    sys->log<System::DEBUG>("[BondedForces] Using %d threads per particle", TPP);
        
  }

  template<class BondType>
  void BondedForces<BondType>::initFixedPoint(){
    int numberParticles = pg->getNumberParticles();    
    sys->log<System::DEBUG>("[BondedForces] Sorting fixed point bonds");
    //Sort in the i index to construct bondStart and bondEnd
    thrust::sort(bondListFP.begin(), bondListFP.end(), BondedForces_ns::BondComp<BondFP>());

    sys->log<System::DEBUG>("[BondedForces] Computing number of particles with bonds");
    thrust::host_vector<BondFP> h_bondList = bondListFP;
    std::set<int> particlesWithBonds;
    fori(0, h_bondList.size()){
      particlesWithBonds.insert(h_bondList[i].i);
    }
    
    int nParticlesWithBonds = particlesWithBonds.size();
    sys->log<System::DEBUG>("[BondedForces] %d particles with fixed point bonds found", nParticlesWithBonds);
    
    bondStartFP.resize(nParticlesWithBonds, nullptr);
    nbondsPerParticleFP.resize(nParticlesWithBonds, 0);
      
    sys->log<System::DEBUG>("[BondedForces] Filling bondStartFP");

    //Fill helper data structures
    int index = 0;
    int nbondsi = 0;
    for(int b = 0; b<nbondsFP; b++){
      int i = h_bondList[b].i;
      int inext;
      if(b<nbondsFP-1) inext = h_bondList[b+1].i;
      else inext = -1;
      nbondsi++;
      if(inext != i){
	if(index == 0)
	  bondStartFP[0] = thrust::raw_pointer_cast(bondListFP.data());    
	else
	  bondStartFP[index] = thrust::raw_pointer_cast(bondListFP.data())+b+1-nbondsi;
	
	nbondsPerParticleFP[index] = nbondsi;

	index++;
	nbondsi = 0;
      }
    }
    }

  namespace BondedForces_ns{

    //I do not really like how this is written now, but it really improves performance...
    
    //This version assigns a block for each particle (thread threadIdx.x handles the bond threadIdx.x of particle blockIdx.x) Much faster when particles have many bonds per particle (>32 maybe)
    template<class Bond, class BondType, int THREADS_PER_BLOCK>
    __global__ void computeBondedForcesBlockPerParticle(real4* __restrict__ force, const real4* __restrict__ pos,
					Bond** __restrict__ bondStart,
					const int* __restrict__ nbondsPerParticle,
					BondType bondForce,
					const int * __restrict__ id2index){
      //This little trick of unionizing the shared memory for the block shared parameters and the blockreduce storage      // does not seem to help much, at least in a GTX980.
      struct Shared{
	Bond const * bondList;
	int nbonds;
	int p;
	real3 posi;
      };
      using BlockReduce = hipcub::BlockReduce<real3, THREADS_PER_BLOCK>;
      __shared__ union{
	Shared info;
	typename BlockReduce::TempStorage temp_storage;
      } shared;
       
      __shared__ int index;

      //Bond list for my particle
      if(threadIdx.x == 0){
	shared.info.bondList = bondStart[blockIdx.x];
	//Number of bonds for my particle
        shared.info.nbonds = nbondsPerParticle[blockIdx.x];
	//My particle index
        shared.info.p = shared.info.bondList[0].i;
	index = id2index[shared.info.p];
	shared.info.posi = make_real3(pos[index]);
      }

      //A block per particle
      //Instead of launching a thread per particle and discarding those without any bond,
      //I store an additional array of size N_particles_with_bonds that contains the indices
      //of the particles that are involved in at least one bond. And only launch N_particles_with_bonds blocks
      
      //Each thread in a block computes the force on particle p due to one (or several) bonds      

      //My local force accumulator
      real3 f = make_real3(real(0.0));
      
      __syncthreads();    
      for(int b = threadIdx.x; b<shared.info.nbonds; b += blockDim.x){
	
	//Read bond info
	const auto bond = shared.info.bondList[b];
	//Bring pos of other particle
	const real3 posj = make_real3(pos[id2index[bond.j]]);
    
	//Compute force
	real3 r12 =  shared.info.posi-posj;
	//Sum force 
	f += bondForce.force(shared.info.p, bond.j, r12, bond.bond_info);
      }

      //Sum the forces of all threads in my block
      real3 ft;
      if(threadIdx.x < shared.info.nbonds){
	ft = BlockReduce(shared.temp_storage).Sum(f);	
      }
      __syncthreads();
      //First thread writes to memory
      if(threadIdx.x == 0){
	force[index] += make_real4(ft);
      }
      
    }


    //This version assigns a thread for each particle (thread i handles all the bonds of particle i), works well when particles have a low number of bonds (<32 per particle)
    template<class Bond, class BondType>
    __global__ void computeBondedForcesThreadPerParticle(real4* __restrict__ force, const real4* __restrict__ pos,
					   Bond** __restrict__ bondStart,
					   const int* __restrict__ nbondsPerParticle,
					   BondType bondForce,
					   const int * __restrict__ id2index, int N){
      int id = blockIdx.x*blockDim.x + threadIdx.x;
      if(id>=N) return;
      
      auto bondList = bondStart[id];
      //Number of bonds for my particle
      const int nbonds = nbondsPerParticle[id];
      //My particle index
      const int p = bondList[0].i;
      const int index = id2index[p];
      const real3 posi = make_real3(pos[index]);
            
      real3 f = make_real3(real(0.0));
      
      for(int b = 0; b<nbonds; b++){
	
	//Read bond info
	const auto bond = bondList[b];
	//Bring pos of other particle
	const real3 posj = make_real3(pos[id2index[bond.j]]);
    
	//Compute force
	real3 r12 =  posi-posj;
      
	f += bondForce.force(p, bond.j, r12, bond.bond_info);
      }
   
      force[index] += make_real4(f);	
    }


    //The same approach could be used for Fixed Point bonds as with p-p bonds.
    template<class Bond, class BondType>
    __global__ void computeBondedForcesFixedPoint(real4* __restrict__ force,
						  const real4* __restrict__ pos,
						  Bond** __restrict__ bondStart,
						  const int* __restrict__ nbondsPerParticle,
						  BondType bondForce,
						  const int * __restrict__ id2index){
      extern __shared__ char shMem[];      
      real3 *forceTotal = (real3*) shMem; //Each thread has a force
      /*
      real4 &posi = *((real4*)(shMem+blockDim.x*sizeof(real4)));
      int &nbonds = *((int*)&posi + sizeof(real4));
      int &p = *((int*)&nbonds + sizeof(int));
      Bond* &bondList = *((Bond**)&p+sizeof(int));
      */
      
      //Bond list for my particle
      const Bond *bondList = bondStart[blockIdx.x];
      //Number of bonds for my particle
      const int nbonds = nbondsPerParticle[blockIdx.x];
      //My particle index
      const int p = bondList[0].i;
      const real3 posi = make_real3(pos[id2index[p]]);

      //A block per particle
      //Instead of launching a thread per particle and discarding those without any bond,
      //I store an additional array of size N_particles_with_bonds that contains the indices
      //of the particles that are involved in at least one bond. And only launch N_particles_with_bonds blocks
      
      //Each thread in a block computes the force on particle p due to one (or several) bonds
      
            
      real3 f = make_real3(real(0.0));          


      //__syncthreads();    
      for(int b = threadIdx.x; b<nbonds; b += blockDim.x){
	
	//Read bond info
	auto bond = bondList[b];	
    
	//Compute force
	real3 r12 =  posi - bond.pos;
      
        

	f += bondForce.force(p,-1, r12, bondList[b].bond_info);
      }

      /*The first thread sums all the contributions*/
      forceTotal[threadIdx.x] = f;
      __syncthreads();
      //TODO Implement a warp reduction
      if(threadIdx.x==0){
	real3 ft = make_real3(0.0f);
	for(int i=0; i<blockDim.x; i++){
	  ft += forceTotal[i];
	}
	/*Write to global memory*/
	force[id2index[p]] += make_real4(ft);
      }

    }     
  }


  //This function chooses which version of the kernel computeBondedForces to use as a function of the number of bonds per particle
  template<class BondType>
  void BondedForces<BondType>::callComputeBondedForces(hipStream_t st){
    auto pos = pd->getPos(access::location::gpu, access::mode::read);
    auto force = pd->getForce(access::location::gpu, access::mode::readwrite);

    auto id2index = pd->getIdOrderedIndices(access::location::gpu);
      
    auto d_bondStart = thrust::raw_pointer_cast(bondStart.data());
    auto d_nbondsPerParticle = thrust::raw_pointer_cast(nbondsPerParticle.data());
    uint Nparticles_with_bonds = bondStart.size();

    
    if(TPP<=32 || Nparticles_with_bonds < 5000){ //Empirical magic numbers, could probably be chosen better
      int Nthreads= 128;
      int Nblocks=Nparticles_with_bonds/Nthreads + ((Nparticles_with_bonds%Nthreads)?1:0);

      BondedForces_ns::computeBondedForcesThreadPerParticle<Bond, BondType>
	<<< Nblocks, Nthreads, 0, st>>>(
	      force.raw(), pos.raw(),
	      d_bondStart, d_nbondsPerParticle,
	      bondForce,
	      id2index, Nparticles_with_bonds);
    }
    else{
      //This is due to cub having blocksize as template parameter, I hate it
      if(TPP>=128){
	BondedForces_ns::computeBondedForcesBlockPerParticle<Bond, BondType, 128>
	  <<<Nparticles_with_bonds, 128, 0, st>>>(
		force.raw(), pos.raw(),
		d_bondStart, d_nbondsPerParticle,
		bondForce, id2index);
      }
      else{
	BondedForces_ns::computeBondedForcesBlockPerParticle<Bond, BondType, 64>
	  <<<Nparticles_with_bonds, 64, 0, st>>>(
	        force.raw(), pos.raw(),
		d_bondStart, d_nbondsPerParticle,
		bondForce, id2index);
      }

    }
  }
  template<class BondType>
  void BondedForces<BondType>::sumForce(hipStream_t st){
    sys->log<System::DEBUG1>("[BondedForces] Computing Forces...");
    if(nbonds>0){
      sys->log<System::DEBUG3>("[BondedForces] Computing Particle-Particle...");
      this->callComputeBondedForces(st);
    }
    if(nbondsFP>0){
      sys->log<System::DEBUG3>("[BondedForces] Computing Fixed-Point...");
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      auto force = pd->getForce(access::location::gpu, access::mode::readwrite);

      auto id2index = pd->getIdOrderedIndices(access::location::gpu);
      
      int numberParticlesWithBonds = bondStartFP.size();
      auto d_bondStart = thrust::raw_pointer_cast(bondStartFP.data());
      auto d_nbondsPerParticle = thrust::raw_pointer_cast(nbondsPerParticleFP.data());      
      BondedForces_ns::computeBondedForcesFixedPoint
	<<<numberParticlesWithBonds,	TPP, TPP*sizeof(real3), st>>>(
								      force.raw(), pos.raw(),
								      d_bondStart,
								      d_nbondsPerParticle,
								      bondForce,
								      id2index);
    }

  
  }
  template<class BondType>
  real BondedForces<BondType>::sumEnergy(){
    return 0;
  }

}