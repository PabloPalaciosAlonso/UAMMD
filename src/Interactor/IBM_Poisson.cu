#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2019. Spectral Poisson solver
 */

#include"IBM_Poisson.cuh"
#include"utils/cufftPrecisionAgnostic.h"
#include"utils/cufftDebug.h"
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>

#include<fstream>
namespace uammd{

  Poisson::Poisson(shared_ptr<ParticleData> pd,
		   shared_ptr<ParticleGroup> pg,
		   shared_ptr<System> sys,
		   Poisson::Parameters par):
    Interactor(pd, pg, sys, "IBM::Poisson"),
    epsilon(par.epsilon),
    box(par.box),
    split(par.split),
    gw(par.gw){

    double h;
    double farFieldGaussianWidth = par.gw;
    if(par.split > 0) farFieldGaussianWidth = sqrt(par.gw*par.gw+1.0/(4.0*par.split*par.split));
    if(par.upsampling>0) h = 1.0/par.upsampling;
    else h = (1.3 - std::min((-log10(par.tolerance))/10.0, 0.9))*farFieldGaussianWidth;
    h = std::min(h, box.boxSize.x/16.0);
    sys->log<System::MESSAGE>("[Poisson] Proposed h: %g", h);
    {
      int3 cellDim = nextFFTWiseSize3D(make_int3(box.boxSize/h));
      grid = Grid(par.box, cellDim);
      h = grid.cellSize.x;
    }
    sys->log<System::MESSAGE>("[Poisson] Selected h: %g", h);
    int ncells = grid.getNumberCells();

    auto kernel = std::make_shared<Kernel>(par.tolerance, farFieldGaussianWidth, h);

    ibm = std::make_shared<IBM<Kernel>>(sys, kernel);

    kernel->support = std::min(kernel->support, grid.cellDim.x/2-2);
    if(split>0){
      long double E=1;
      long double r = farFieldGaussianWidth;
      while(abs(E)>par.tolerance){
	r+=0.001l;
	E = 1.0l/(4.0l*M_PIl*epsilon*r)*(erf(r/(2.0l*gw))- erf(r/sqrt(4.0l*gw*gw+1/(split*split))));
      }
      nearFieldCutOff = std::min(r, box.boxSize.x/1.999l);
    }	
    
    sys->log<System::MESSAGE>("[Poisson] tolerance: %g", par.tolerance);
    sys->log<System::MESSAGE>("[Poisson] support: %d", kernel->support);
    sys->log<System::MESSAGE>("[Poisson] epsilon: %g", epsilon);
    sys->log<System::MESSAGE>("[Poisson] Gaussian source width: %g", par.gw);
    sys->log<System::MESSAGE>("[Poisson] cells: %d %d %d", grid.cellDim.x, grid.cellDim.y, grid.cellDim.z);
    sys->log<System::MESSAGE>("[Poisson] box size: %g %g %g", box.boxSize.x, box.boxSize.y, box.boxSize.z);
    if(par.split> 0){
      sys->log<System::MESSAGE>("[Poisson] Ewald split mode enabled");
      sys->log<System::MESSAGE>("[Poisson] split: %g", par.split);
      sys->log<System::MESSAGE>("[Poisson] Far field width: %g, (%g times original width)",
				farFieldGaussianWidth, 1/par.gw*sqrt(1/(4*par.split*par.split)+par.gw*par.gw));
      sys->log<System::MESSAGE>("[Poisson] Near field cut off: %g", nearFieldCutOff);
    }
    CudaSafeCall(hipStreamCreate(&st));
    CudaCheckError();
    initCuFFT();
  }

  Poisson::~Poisson(){
    CudaSafeCall(hipStreamDestroy(st));
  }
  void Poisson::initCuFFT(){

    CufftSafeCall(hipfftCreate(&cufft_plan_forward));
    CufftSafeCall(hipfftSetAutoAllocation(cufft_plan_forward, 0));

    CufftSafeCall(hipfftCreate(&cufft_plan_inverse));
    CufftSafeCall(hipfftSetAutoAllocation(cufft_plan_inverse, 0));

    //Required storage for the plans
    size_t cufftWorkSizef = 0, cufftWorkSizei;//f = 0, cufftWorkSizeie = 0;
    /*Set up cuFFT*/
    //This sizes have to be reversed according to the cufft docs
    int3 cdtmp = {grid.cellDim.z, grid.cellDim.y, grid.cellDim.x};
    int3 inembed = {grid.cellDim.z, grid.cellDim.y, grid.cellDim.x};

    //A single forward fft for the charges
    CufftSafeCall(hipfftMakePlan3d(cufft_plan_forward, cdtmp.x, cdtmp.y, cdtmp.z,
				  CUFFT_Real2Complex<real>::value,
				  &cufftWorkSizef));

    sys->log<System::DEBUG>("[BDHI::Poisson] cuFFT grid size: %d %d %d", cdtmp.x, cdtmp.y, cdtmp.z);
    //Force and energy in a single transform
    /*I want to make four 3D FFTs, each one using one of the three interleaved coordinates*/
    CufftSafeCall(hipfftMakePlanMany(cufft_plan_inverse,
				    3, &cdtmp.x, /*Three dimensional FFT*/
				    &inembed.x,
				    /*Each FFT starts in 1+previous FFT index. FFTx in 0*/
				    4, 1, //Each element separated by four others fx0 fy0 fz0 e0 fx1 fy1 fz1 e1...
				    &inembed.x,
				    4, 1,
				    CUFFT_Complex2Real<real>::value, 4,
				    &cufftWorkSizei));

    /*Allocate cuFFT work area*/
    size_t cufftWorkSize = std::max(cufftWorkSizef, cufftWorkSizei)+10;
    size_t free_mem, total_mem;
    CudaSafeCall(hipMemGetInfo(&free_mem, &total_mem));

    sys->log<System::DEBUG>("[BDHI::Poisson] Necessary work space for cuFFT: %s, available: %s, total: %s",
			    printUtils::prettySize(cufftWorkSize).c_str(),
			    printUtils::prettySize(free_mem).c_str(),
			    printUtils::prettySize(total_mem).c_str());

    if(free_mem<cufftWorkSize){
      sys->log<System::CRITICAL>("[BDHI::Poisson] Not enough memory in device to allocate cuFFT free %s, needed: %s!!",
				 printUtils::prettySize(free_mem).c_str(),
				 printUtils::prettySize(cufftWorkSize).c_str());
    }

    cufftWorkArea.resize(cufftWorkSize);
    auto d_cufftWorkArea = thrust::raw_pointer_cast(cufftWorkArea.data());

    CufftSafeCall(hipfftSetWorkArea(cufft_plan_forward, (void*)d_cufftWorkArea));
    CufftSafeCall(hipfftSetWorkArea(cufft_plan_inverse, (void*)d_cufftWorkArea));
  }

  namespace Poisson_ns{

    struct NearFieldEnergyTransverser{
      using returnInfo = real;
      
      NearFieldEnergyTransverser(real* energy_ptr, real* charge,
			   real ep, real sp, real gw, Box box):
	energy_ptr(energy_ptr), charge(charge),
	epsilon(ep), split(sp), gw(gw), box(box){}

      inline __device__ returnInfo zero() const{ return 0.0f;}
      inline __device__ real getInfo(int pi) const{ return charge[pi];}
      
      inline __device__ returnInfo compute(const real4 &pi, const real4 &pj, real chargei, real chargej) const{
	real E = 0;
	real3 rij = box.apply_pbc(make_real3(pj)-make_real3(pi));
	real r2 = dot(rij, rij);
	if(r2>gw*gw*gw*gw){
	  real r = sqrt(r2);
	  E = chargei*(1.0/(4.0*M_PI*epsilon*r)*(erf(r/(2*gw)) - erf(r/sqrt(4*gw*gw+1/(split*split)))));
	}
	else{
	  const real pi32 = pow(M_PI,1.5);
	  const real gw2 = gw*gw;
	  const real invsp2 = 1.0/(split*split);
	  const real selfterm = 1.0/(4*pi32*gw) - 1.0/(2*pi32*sqrt(4*gw2+invsp2));
	  const real r2term = 1.0/(6.0*pi32*pow(4.0*gw2 + invsp2, 1.5)) - 1.0/(48.0*pi32*gw2*gw);
	  const real r4term = 1.0/(640.0*pi32*gw2*gw2*gw) - 1.0/(20.0*pi32*pow(4*gw2+invsp2,2.5));
	  E = chargei/epsilon*(selfterm+r2*r2term + r2*r2*r4term);
	}
	return E;
      }
      inline __device__ void accumulate(returnInfo &total, const returnInfo &current) const {total += current;}
      inline __device__ void set(uint pi, const returnInfo &total) const {energy_ptr[pi] += total;}
    private:
      real* energy_ptr;
      real* charge;
      real epsilon, split, gw;
      Box box;
    };

    struct NearFieldForceTransverser{
      using returnInfo = real3;
      
      NearFieldForceTransverser(real4* force_ptr, real* charge,
				 real ep, real sp, real gw, Box box):
	force_ptr(force_ptr), charge(charge),
	epsilon(ep), split(sp), gw(gw), box(box){}

      inline __device__ returnInfo zero() const{ return returnInfo();}
      inline __device__ real getInfo(int pi) const{ return charge[pi];}
      
      inline __device__ returnInfo compute(const real4 &pi, const real4 &pj, real chargei, real chargej) const{

	real3 rij = box.apply_pbc(make_real3(pj)-make_real3(pi));
	real r2 = dot(rij, rij);
	real r = sqrt(r2);
	real gw2 = gw*gw;
	real newgw = sqrt(gw2+1/(4.0*split*split));
	real newgw2 = newgw*newgw;
	real fmod = 0;
	if(r2>gw*gw*gw*gw){
	  real invrterm = exp(-0.25*r2/newgw2)/sqrt(M_PI*newgw2) - exp(-0.25*r2/gw2)/sqrt(M_PI*gw2);
	  real invr2term = erf(0.5*r/newgw) - erf(0.5*r/gw);
	  
	  fmod += 1/(4*M_PI)*( invrterm/r - invr2term/r2);
	}
	else if (r2>0){
	  const real pi32 = pow(M_PI, 1.5);
	  real rterm = 1/(24*pi32)*(1.0/(gw2*gw) - 1/(newgw2*newgw));
	  real r3term = 1/(160*pi32)*(1.0/(newgw2*newgw2*newgw) - 1.0/(gw2*gw2*gw));
	  fmod += r*rterm+r2*r*r3term;
	}
	if(r2>0) return chargei/epsilon*fmod*rij/r;
	else return real3();
      }
      inline __device__ void accumulate(returnInfo &total, const returnInfo &current) const {total += current;}
      inline __device__ void set(uint pi, const returnInfo &total) const {force_ptr[pi] += make_real4(total);}
    private:
      real4* force_ptr;
      real* charge;
      real epsilon, split, gw;
      Box box;
    };


  }
  void Poisson::farField(hipStream_t st){
    try{
      gridCharges.resize(2*grid.cellDim.y*grid.cellDim.z*(grid.cellDim.x+1));
      gridForceEnergy.resize(grid.getNumberCells());
      thrust::fill(gridCharges.begin(), gridCharges.end(),
		   std::iterator_traits<decltype(gridCharges.begin())>::value_type());
    }
    catch(thrust::system_error &e){
      sys->log<System::CRITICAL>("[Poisson] Thrust could not reset grid data with error &s", e.what());
    }
    sys->log<System::DEBUG2>("[Poisson] Far field computation");
    spreadCharges();
    forwardTransformCharge();
    convolveFourier();
    inverseTransform();
    interpolateFields();

  }
  void Poisson::sumForce(hipStream_t st){
    sys->log<System::DEBUG2>("[Poisson] Sum Force");

    farField(st);
    
    if(split>0){
      sys->log<System::DEBUG2>("[Poisson] Near field force computation");
      if(!nl) nl = std::make_shared<NeighbourList>(pd, pg, sys);

      nl->updateNeighbourList(box, nearFieldCutOff, st);
      auto force = pd->getForce(access::location::gpu, access::mode::readwrite);
      auto charge = pd->getCharge(access::location::gpu, access::mode::read);
      {
	auto tr = Poisson_ns::NearFieldForceTransverser(force.begin(), charge.begin(),
							epsilon, split, gw, box);
	nl->transverseList(tr, st);
      }
    }



  }

  real Poisson::sumEnergy(){
    sys->log<System::DEBUG2>("[Poisson] Sum Energy");
    hipStream_t st = 0;
    farField(st);
    if(split>0){

      sys->log<System::DEBUG2>("[Poisson] Near field energy computation");
      sys->log<System::WARNING>("[Poisson] Not subtracting adding phi(0,0,0)");
      if(!nl) nl = std::make_shared<NeighbourList>(pd, pg, sys);

      nl->updateNeighbourList(box, nearFieldCutOff, st);
      auto energy = pd->getEnergy(access::location::gpu, access::mode::readwrite);
      auto charge = pd->getCharge(access::location::gpu, access::mode::read);
      {
      auto tr = Poisson_ns::NearFieldEnergyTransverser(energy.begin(), charge.begin(),
						 epsilon, split, gw, box);
      nl->transverseList(tr, st);
      }
    }

    return 0;
  }

  namespace Poisson_ns{
    using cufftComplex3 = Poisson::cufftComplex3;
    using cufftComplex4 = Poisson::cufftComplex4;
    using hipfftComplex = Poisson::hipfftComplex;

    template<class vec3>
    inline __device__ vec3 cellToWaveNumber(const int3 &cell, const int3 &cellDim, const vec3 &L){
      const vec3 pi2invL = (real(2.0)*real(M_PI))/L;
      vec3 k = {cell.x*pi2invL.x,
		cell.y*pi2invL.y,
		cell.z*pi2invL.z};
      if(cell.x >= (cellDim.x/2+1)) k.x -= real(cellDim.x)*pi2invL.x;
      if(cell.y >= (cellDim.y/2+1)) k.y -= real(cellDim.y)*pi2invL.y;
      if(cell.z >= (cellDim.z/2+1)) k.z -= real(cellDim.z)*pi2invL.z;
      return k;
    }

    __global__ void chargeFourier2ForceAndEnergy(hipfftComplex* gridCharges,
						 cufftComplex4* gridForceEnergy,
						 real epsilon,
						 Grid grid){
      int3 cell;
      cell.x= blockIdx.x*blockDim.x + threadIdx.x;
      cell.y= blockIdx.y*blockDim.y + threadIdx.y;
      cell.z= blockIdx.z*blockDim.z + threadIdx.z;

      if(cell.x>=grid.cellDim.x/2+2) return;
      if(cell.y>=grid.cellDim.y) return;
      if(cell.z>=grid.cellDim.z) return;

      const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);
      const real k2 = dot(k,k);

      if(k2 == 0){
	gridForceEnergy[0] = cufftComplex4();
	return;
      }

      const real B = real(1.0)/(k2*epsilon*grid.getNumberCells());

      const int i_icell = cell.x + (cell.y + cell.z*grid.cellDim.y)*(grid.cellDim.x/2+1);

      const hipfftComplex fk = gridCharges[i_icell];
      cufftComplex4 force = cufftComplex4();

      bool nyquist = false;
      { //Is the current wave number a nyquist point?
       	bool isXnyquist = (cell.x == grid.cellDim.x - cell.x) && (grid.cellDim.x%2 == 0);
       	bool isYnyquist = (cell.y == grid.cellDim.y - cell.y) && (grid.cellDim.y%2 == 0);
       	bool isZnyquist = (cell.z == grid.cellDim.z - cell.z) && (grid.cellDim.z%2 == 0);

       	nyquist =  (isXnyquist && cell.y==0   && cell.z==0)  or  //1
       	  (isXnyquist && isYnyquist  && cell.z==0)  or  //2
       	  (cell.x==0  && isYnyquist  && cell.z==0)  or  //3
       	  (isXnyquist && cell.y==0   && isZnyquist) or  //4
       	  (cell.x==0  && cell.y==0   && isZnyquist) or  //5
       	  (cell.x==0  && isYnyquist  && isZnyquist) or  //6
       	  (isXnyquist && isYnyquist  && isZnyquist);    //7
      }

      if(nyquist) force = cufftComplex4();
      else{
	force.x.x = k.x*fk.y*B; force.x.y = -k.x*fk.x*B;
	force.y.x = k.y*fk.y*B; force.y.y = -k.y*fk.x*B;
	force.z.x = k.z*fk.y*B; force.z.y = -k.z*fk.x*B;
      }
      force.w = fk*B;
      // force.w.x = abs(fk.x*fk.x -fk.y*fk.y)*B; //Energy
      // force.w.y = abs(2*fk.x*fk.y)*B; //Energy

      const int o_icell = grid.getCellIndex(cell);
      gridForceEnergy[o_icell] = force;
    }

  }
  void Poisson::spreadCharges(){
    sys->log<System::DEBUG2>("[Poisson] Spreading charges");
    int numberParticles = pg->getNumberParticles();
    auto pos = pd->getPos(access::location::gpu, access::mode::read);
    auto charges = pd->getCharge(access::location::gpu, access::mode::read);

    real* d_gridCharges = (real*)thrust::raw_pointer_cast(gridCharges.data());

    ibm->spread(pos.begin(), charges.begin(), d_gridCharges, grid, numberParticles, st);
    CudaCheckError();
  }
  void Poisson::forwardTransformCharge(){
    CufftSafeCall(hipfftSetStream(cufft_plan_forward, st));
    auto d_gridCharges = thrust::raw_pointer_cast(gridCharges.data());
    auto d_gridChargesFourier = thrust::raw_pointer_cast(gridCharges.data())+grid.getNumberCells();
    sys->log<System::DEBUG2>("[Poisson] Taking grid to wave space");
    {
      auto cufftStatus =
	cufftExecReal2Complex<real>(cufft_plan_forward,
				    (hipfftReal*)d_gridCharges,
				    (hipfftComplex*)d_gridChargesFourier);
      if(cufftStatus != HIPFFT_SUCCESS){
	sys->log<System::CRITICAL>("[Poisson] Error in forward CUFFT");
      }
    }
  }

  void Poisson::inverseTransform(){
    sys->log<System::DEBUG2>("[Poisson] Force to real space");
    CufftSafeCall(hipfftSetStream(cufft_plan_inverse, st));
    auto d_gridForceEnergy = thrust::raw_pointer_cast(gridForceEnergy.data());
    auto d_gridForceEnergyFourier = thrust::raw_pointer_cast(gridForceEnergy.data());

    {
      auto cufftStatus =
	cufftExecComplex2Real<real>(cufft_plan_inverse,
				    (hipfftComplex*)d_gridForceEnergyFourier,
				    (hipfftReal*)d_gridForceEnergy);
      if(cufftStatus != HIPFFT_SUCCESS){
	sys->log<System::CRITICAL>("[Poisson] Error in inverse CUFFT");
      }
    }
    CudaCheckError();
  }


  void Poisson::convolveFourier(){

    auto d_gridChargesFourier = thrust::raw_pointer_cast(gridCharges.data())+grid.getNumberCells();

    cufftComplex4* d_gridForceEnergyFourier = thrust::raw_pointer_cast(gridForceEnergy.data());

    sys->log<System::DEBUG2>("[Poisson] Wave space convolution");
    {
      dim3 NthreadsCells = dim3(8,8,8);
      dim3 NblocksCells;
      {
	int ncellsx = grid.cellDim.x/2+1;
	NblocksCells.x= (ncellsx/NthreadsCells.x + ((ncellsx%NthreadsCells.x)?1:0));
	NblocksCells.y= grid.cellDim.y/NthreadsCells.y + ((grid.cellDim.y%NthreadsCells.y)?1:0);
	NblocksCells.z= grid.cellDim.z/NthreadsCells.z + ((grid.cellDim.z%NthreadsCells.z)?1:0);
      }

      Poisson_ns::chargeFourier2ForceAndEnergy<<<NblocksCells, NthreadsCells, 0, st>>>
	((hipfftComplex*) d_gridChargesFourier,
	 d_gridForceEnergyFourier,
	 epsilon,
	 grid);
    }
    CudaCheckError();

  }

  namespace Poisson_ns{

    struct toReal4{
      __device__ real4 operator()(real3 a){
	return make_real4(a);
      }
    };

    struct Zip2Real4{

      real4* force;
      real* energy;
      int i;

      Zip2Real4(real4* f, real* e):force(f), energy(e), i(-1){}
      __device__ Zip2Real4 operator()(int ai){
	this->i = ai;
	return *this;
      }

      __device__ void operator += (real4 fande){
	force[i] += make_real4(fande.x, fande.y, fande.z, 0);
	energy[i] += fande.w;
      }
    };

  }
  void Poisson::interpolateFields(){
    sys->log<System::DEBUG2>("[Poisson] Interpolating forces and energies");

    int numberParticles = pg->getNumberParticles();
    auto pos = pd->getPos(access::location::gpu, access::mode::read);
    auto forces = pd->getForce(access::location::gpu, access::mode::readwrite);
    auto energies = pd->getEnergy(access::location::gpu, access::mode::readwrite);

    real4* d_gridForcesEnergies = (real4*)thrust::raw_pointer_cast(gridForceEnergy.data());
    //Transform real4 grid data into separated force and energy particle arrays
    auto f_tr = thrust::make_transform_iterator(thrust::make_counting_iterator<int>(0),
						Poisson_ns::Zip2Real4(forces.begin(),
								      energies.begin())
						);

    ibm->gather(pos.begin(),
   		f_tr,
   		d_gridForcesEnergies,
   		grid, numberParticles, st);


  }

}