#include "hip/hip_runtime.h"
/*Raul P. Pelaez 2018. Force Coupling Method BDHI Module.
  See BDHI_FCM.cuh for information.
*/
#include"BDHI_FCM.cuh"
#include"utils/GPUUtils.cuh"
#include"third_party/saruprng.cuh"
#include<vector>
#include<algorithm>
#include<fstream>
#include"utils/debugTools.h"
#include"utils/cufftDebug.h"
#include"utils/cxx_utils.h"
#include"third_party/type_names.h"
#include<fstream>
namespace uammd{
  namespace BDHI{

    FCM::FCM(shared_ptr<ParticleData> pd,
	     shared_ptr<ParticleGroup> pg,
	     shared_ptr<System> sys,
	     Parameters par):
      pd(pd), pg(pg), sys(sys),
      dt(par.dt),
      temperature(par.temperature),
      viscosity(par.viscosity),
      box(par.box), grid(par.box, int3()){

      seed = sys->rng().next();
      sys->log<System::MESSAGE>("[BDHI::FCM] Initialized");
      sys->log<System::MESSAGE>("[BDHI::FCM] Using kernel: %s", type_name<Kernel>().c_str());
      if(box.boxSize.x == real(0.0) && box.boxSize.y == real(0.0) && box.boxSize.z == real(0.0)){
	sys->log<System::CRITICAL>("[BDHI::FCM] Box of size zero detected, cannot work without a box! (make sure a box parameter was passed)");
      }

      int numberParticles = pg->getNumberParticles();

      fac = par.fac;
      int3 cellDim;
      if(par.cells.x<=0){
	if(par.hydrodynamicRadius<=0)
	  sys->log<System::CRITICAL>("[BDHI::FCM] I need an hydrodynamic radius if cell dimensions are not provided!");
	real h = par.hydrodynamicRadius;
	cellDim = nextFFTWiseSize3D(make_int3(box.boxSize/h));
      }
      else{
        cellDim = par.cells;
      }
      grid = Grid(box, cellDim);

      auto kernel = std::make_shared<Kernel>(grid.cellSize, par.tolerance);
      ibm = std::make_shared<IBM<Kernel>>(sys, kernel, grid);

      double rh = this->getHydrodynamicRadius();

      // {
      //  	std::ofstream out("kern.dat");

      //  	int Nt = 10000;
      //  	double rmax = 6;

      //  	double r = 0;
      //  	double dr = rmax/Nt;
      //  	fori(0,Nt){
      //  	  out<<r<<" "<<kernel->phi(r)<<"\n";
      //  	  r+=dr;

      //  	}
      //  	out<<std::endl;
      //  }

      // {
      //  	 std::ofstream out("kern2.dat");

      //  	 int Nt = 10000;
      //  	 double rmax = 6;

      //  	 double r = 0;
      //  	 double dr = rmax/Nt;
      //  	 auto k2 = std::make_shared<IBM_kernels::GaussianFlexible::sixPoint>(grid.cellSize, 1e-10);
      // 	 //auto k2 = std::make_shared<IBM_kernels::PeskinKernel::threePoint>(grid.cellSize, 1e-10);
      // 	 //auto k2 = std::make_shared<IBM_kernels::PeskinKernel::fourPoint>(grid.cellSize, 1e-10);

      //  	 fori(0,Nt){
      //  	   out<<r<<" "<<k2->phi(r)<<"\n";
      //  	   r+=dr;

      //  	 }
      //  	 out<<std::endl;
      //  }
       // exit(1);


      //Try to set the closest rh possible
      if(par.cells.x<=0){
       	double fac = rh/par.hydrodynamicRadius;
       	double h = grid.cellSize.x/fac;
       	cellDim = nextFFTWiseSize3D(make_int3(box.boxSize/h));
       	grid = Grid(box, cellDim);
       	kernel = std::make_shared<Kernel>(grid.cellSize, par.tolerance);
	ibm = std::make_shared<IBM<Kernel>>(sys, kernel, grid);
       	rh = this->getHydrodynamicRadius();
      }
      long double M0 = this->getSelfMobility();

      sys->log<System::MESSAGE>("[BDHI::FCM] Closest possible hydrodynamic radius: %g (%g requested)", rh, par.hydrodynamicRadius);
      sys->log<System::MESSAGE>("[BDHI::FCM] Self mobility: %g", (double)M0);

      if(box.boxSize.x != box.boxSize.y || box.boxSize.y != box.boxSize.z || box.boxSize.x != box.boxSize.z){
	sys->log<System::WARNING>("[BDHI::FCM] Self mobility will be different for non cubic boxes!");
      }

      sys->log<System::MESSAGE>("[BDHI::FCM] Box Size: %g %g %g", grid.box.boxSize.x, grid.box.boxSize.y, grid.box.boxSize.z);

      sys->log<System::MESSAGE>("[BDHI::FCM] Grid dimensions: %d %d %d", grid.cellDim.x, grid.cellDim.y, grid.cellDim.z);
      sys->log<System::MESSAGE>("[BDHI::FCM] Interpolation kernel support: %g rh max distance, %d cells total", kernel->support*0.5*grid.cellSize.x/rh, kernel->support);

      sys->log<System::MESSAGE>("[BDHI::FCM] h: %g %g %g", grid.cellSize.x, grid.cellSize.y, grid.cellSize.z);
      sys->log<System::MESSAGE>("[BDHI::FCM] Cell volume: %e", grid.cellSize.x*grid.cellSize.y*grid.cellSize.z);
      sys->log<System::MESSAGE>("[BDHI::FCM] Requested kernel tolerance: %g", par.tolerance);

      if(kernel->support >= grid.cellDim.x or
	 kernel->support >= grid.cellDim.y or
	 kernel->support >= grid.cellDim.z)
	sys->log<System::ERROR>("[BDHI::FCM] Kernel support is too big, try lowering the tolerance or increasing the box size!.");

      CudaSafeCall(hipStreamCreate(&stream));
      CudaSafeCall(hipStreamCreate(&stream2));

      /*The quantity spreaded to the grid in real or wave space*/
      /*The layout of this array is
	fx000, fy000, fz000, fx001, fy001, fz001..., fxnnn, fynnn, fznnn. n=ncells-1
	When used in real space each f is a real number, whereas in wave space each f will be a complex number.
	See cufftC2R of R2C in place in Mdot_far
      */
      /*Can be Force when spreading particles to the grid and
	velocities when interpolating from the grid to the particles*/
      int ncells = grid.cellDim.x*grid.cellDim.y*grid.cellDim.z;
      gridVelsFourier.resize(3*ncells, hipfftComplex());

      initCuFFT();

      CudaSafeCall(hipDeviceSynchronize());
      CudaCheckError();
    }

    void FCM::initCuFFT(){
      CufftSafeCall(hipfftCreate(&cufft_plan_forward));
      CufftSafeCall(hipfftCreate(&cufft_plan_inverse));

      /*I will be handling workspace memory*/
      CufftSafeCall(hipfftSetAutoAllocation(cufft_plan_forward, 0));
      CufftSafeCall(hipfftSetAutoAllocation(cufft_plan_inverse, 0));

      //Required storage for the plans
      size_t cufftWorkSizef = 0, cufftWorkSizei = 0;
      /*Set up cuFFT*/
      //This sizes have to be reversed according to the cufft docs
      int3 cdtmp = {grid.cellDim.z, grid.cellDim.y, grid.cellDim.x};
      int3 inembed = {grid.cellDim.z, grid.cellDim.y, grid.cellDim.x};
      /*I want to make three 3D FFTs, each one using one of the three interleaved coordinates*/
      CufftSafeCall(hipfftMakePlanMany(cufft_plan_forward,
				      3, &cdtmp.x, /*Three dimensional FFT*/
				      &inembed.x,
				      /*Each FFT starts in 1+previous FFT index. FFTx in 0*/
				      3, 1, //Each element separated by three others x0 y0 z0 x1 y1 z1...
				      /*Same format in the output*/
				      &inembed.x,
				      3, 1,
				      /*Perform 3 direct Batched FFTs*/
				      HIPFFT_R2C, 3,
				      &cufftWorkSizef));

      sys->log<System::DEBUG>("[BDHI::FCM] cuFFT grid size: %d %d %d", cdtmp.x, cdtmp.y, cdtmp.z);
      /*Same as above, but with C2R for inverse FFT*/
      CufftSafeCall(hipfftMakePlanMany(cufft_plan_inverse,
				      3, &cdtmp.x, /*Three dimensional FFT*/
				      &inembed.x,
				      /*Each FFT starts in 1+previous FFT index. FFTx in 0*/
				      3, 1, //Each element separated by three others x0 y0 z0 x1 y1 z1...
				      &inembed.x,
				      3, 1,
				      /*Perform 3 inverse batched FFTs*/
				      HIPFFT_C2R, 3,
				      &cufftWorkSizei));

      /*Allocate cuFFT work area*/
      size_t cufftWorkSize = std::max(cufftWorkSizef, cufftWorkSizei)+10;
      size_t free_mem, total_mem;
      CudaSafeCall(hipMemGetInfo(&free_mem, &total_mem));

      sys->log<System::DEBUG>("[BDHI::FCM] Necessary work space for cuFFT: %s, available: %s, total: %s",
			      printUtils::prettySize(cufftWorkSize).c_str(),
			      printUtils::prettySize(free_mem).c_str(),
			      printUtils::prettySize(total_mem).c_str());

      if(free_mem<cufftWorkSize){
	sys->log<System::CRITICAL>("[BDHI::FCM] Not enough memory in device to allocate cuFFT free %s, needed: %s!!",
				   printUtils::prettySize(free_mem).c_str(),
				   printUtils::prettySize(cufftWorkSize).c_str());
      }

      cufftWorkArea.resize(cufftWorkSize);
      auto d_cufftWorkArea = thrust::raw_pointer_cast(cufftWorkArea.data());

      CufftSafeCall(hipfftSetWorkArea(cufft_plan_forward, (void*)d_cufftWorkArea));
      CufftSafeCall(hipfftSetWorkArea(cufft_plan_inverse, (void*)d_cufftWorkArea));
    }

    FCM::~FCM(){
      CudaCheckError();
      CudaSafeCall(hipDeviceSynchronize());
      CufftSafeCall(hipfftDestroy(cufft_plan_inverse));
      CufftSafeCall(hipfftDestroy(cufft_plan_forward));
      CudaSafeCall(hipStreamDestroy(stream));
      CudaSafeCall(hipStreamDestroy(stream2));
      CudaCheckError();
    }

    //Compute M·v = Mw·v
    template<typename vtype>
    void FCM::Mdot(real3 *Mv, vtype *v, hipStream_t st){
      sys->log<System::DEBUG1>("[BDHI::FCM] Mdot....");
      {
	int numberParticles = pg->getNumberParticles();
	int BLOCKSIZE = 128;
	int Nthreads = BLOCKSIZE<numberParticles?BLOCKSIZE:numberParticles;
	int Nblocks  =  numberParticles/Nthreads +  ((numberParticles%Nthreads!=0)?1:0);

	fillWithGPU<<<Nblocks, Nthreads, 0, st>>>(Mv, make_real3(0.0), numberParticles);
      }
      Mdot_far<vtype>(Mv, v, st);

    }
    namespace FCM_ns{

      using cufftComplex3 = FCM::cufftComplex3;

      /*This function takes a node index and returns the corresponding wave number*/
      template<class vec3>
      inline __device__ vec3 cellToWaveNumber(const int3 &cell, const int3 &cellDim, const vec3 &L){
	const vec3 pi2invL = (real(2.0)*real(M_PI))/L;
	/*My wave number*/
	vec3 k = {cell.x*pi2invL.x,
		  cell.y*pi2invL.y,
		  cell.z*pi2invL.z};
	/*Be careful with the conjugates*/
	/*Remember that FFT stores wave numbers as K=0:N/2+1:-N/2:-1 */
	if(cell.x >= (cellDim.x+1)/2) k.x -= real(cellDim.x)*pi2invL.x;
	if(cell.y >= (cellDim.y+1)/2) k.y -= real(cellDim.y)*pi2invL.y;
	if(cell.z >= (cellDim.z+1)/2) k.z -= real(cellDim.z)*pi2invL.z;
	return k;
      }

      /*Apply the projection operator to a wave number with a certain complex factor.
	res = (I-\hat{k}^\hat{k})·factor*/

      inline __device__ cufftComplex3 projectFourier(const real3 &k, const cufftComplex3 &factor){
	const real invk2 = real(1.0)/dot(k,k);

	cufftComplex3 res;
	{//Real part
	  const real3 fr = make_real3(factor.x.x, factor.y.x, factor.z.x);
	  const real kfr = dot(k,fr)*invk2;
	  const real3 vr = (fr-k*kfr);
	  res.x.x = vr.x;
	  res.y.x = vr.y;
	  res.z.x = vr.z;
	}
	{//Imaginary part
	  const real3 fi = make_real3(factor.x.y, factor.y.y, factor.z.y);
	  const real kfi = dot(k,fi)*invk2;
	  const real3 vi = (fi-k*kfi);
	  res.x.y = vi.x;
	  res.y.y = vi.y;
	  res.z.y = vi.z;
	}
	return res;
      }

      /*Scales fourier transformed forces in the regular grid to obtain velocities,
	(Mw·F)_deterministic = σ·St·FFTi·B·FFTf·S·F
	 Input: gridForces = FFTf·S·F
	 Output:gridVels = B·FFTf·S·F -> B \propto (I-k^k/|k|^2)
       */
      /*A thread per fourier node*/
      __global__ void forceFourier2Vel(cufftComplex3 * gridForces, /*Input array*/
				       cufftComplex3 * gridVels, /*Output array, can be the same as input*/
				       real vis,
				       Grid grid/*Grid information and methods*/
				       ){
	/*Get my cell*/
	int3 cell;
	cell.x= blockIdx.x*blockDim.x + threadIdx.x;
	cell.y= blockIdx.y*blockDim.y + threadIdx.y;
	cell.z= blockIdx.z*blockDim.z + threadIdx.z;
	/*Only the first half of the innermost dimension is stored, the rest is redundant*/
	if(cell.x>=grid.cellDim.x/2+2) return;
	if(cell.y>=grid.cellDim.y) return;
	if(cell.z>=grid.cellDim.z) return;

	const int icell = grid.getCellIndex(cell);
	if(icell == 0){
	  gridVels[0] = {0,0, 0,0, 0,0};
	  return;
	}
	const int ncells = grid.getNumberCells();
	const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);
	const real invk2 = real(1.0)/dot(k,k);
	/*Get my scaling factor B, Fourier representation of FCM kernel*/
	const real B = invk2/(vis*real(ncells));
	cufftComplex3 factor = gridForces[icell];

	factor.x *= B;
	factor.y *= B;
	factor.z *= B;

	/*Store vel in global memory, note that this is overwritting any previous value in gridVels*/
	gridVels[icell] = projectFourier(k, factor);
      }

      /*Computes the long range stochastic velocity term
	Mw·F + sqrt(Mw)·dWw = σ·St·FFTi·B·FFTf·S·F+ √σ·St·FFTi·√B·dWw =
	= σ·St·FFTi( B·FFTf·S·F + 1/√σ·√B·dWw)
	This kernel gets v_k = gridVelsFourier = B·FFtt·S·F as input and adds 1/√σ·√B(k)·dWw.
	Keeping special care that v_k = v*_{N-k}, which implies that dWw_k = dWw*_{N-k}
	See eq. 30 in [1].
	Launch a thread per cell grid/fourier node
      */
      __global__ void fourierBrownianNoise(/*Values of vels on each cell*/
					   cufftComplex3 *__restrict__ gridVelsFourier,
					   Grid grid, /*Grid parameters. Size of a cell, number of cells...*/
					   real prefactor,/* sqrt(2·T/(dt·dV))*/
					   real vis,
					   //Parameters to seed the RNG
					   ullint seed,
					   ullint step
					   ){
	/*Get my cell*/
	int3 cell;
	cell.x= blockIdx.x*blockDim.x + threadIdx.x;
	cell.y= blockIdx.y*blockDim.y + threadIdx.y;
	cell.z= blockIdx.z*blockDim.z + threadIdx.z;
	/*This indesx is computed here to use it as a seed for the RNG*/
	int icell = grid.getCellIndex(cell);
	/*cuFFT R2C and C2R only store half of the innermost dimension, the one that varies the fastest

	  The input of R2C is real and the output of C2R is real.
	  The only way for this to be true is if v_k={i,j,k} = v*_k{N-i, N-j, N-k}

	  So the conjugates are redundant and the is no need to compute them nor store them except on two exceptions.
	  In this scheme, the only cases in which v_k and v_{N-k} are stored are:
	     1- When the innermost dimension coordinate is 0.
	     2- When the innermost dimension coordinate is N/2 and N is even.
	*/
	/*Only compute the first half of the innermost dimension*/
	if(2*cell.x >= grid.cellDim.x+1) return;
	if(cell.y >= grid.cellDim.y) return;
	if(cell.z >= grid.cellDim.z) return;

	const int ncells = grid.getNumberCells();
	/*K=0 is not added, no stochastic motion is added to the center of mass*/
	if((cell.x == 0 and cell.y == 0 and cell.z == 0) or
	   /*These terms will be computed along its conjugates*/
	   /*These are special because the conjugate of k_i=0 is k_i=N_i,
	     which is not stored and therfore must not be computed*/
	   (cell.x==0 and cell.y == 0 and 2*cell.z >= grid.cellDim.z+1) or
	   (cell.x==0 and 2*cell.y >= grid.cellDim.y+1)) return;

	/*Compute gaussian complex noise dW,
	  std = prefactor -> ||z||^2 = <x^2>/sqrt(2)+<y^2>/sqrt(2) = prefactor*/
	/*A complex random number for each direction*/
	cufftComplex3 noise;
	{
	  //Uncomment to use uniform numbers instead of gaussian
	  Saru saru(icell, step, seed);
	  const real complex_gaussian_sc = real(0.707106781186547)*prefactor; //1/sqrt(2)
	  //const real sqrt32 = real(1.22474487139159)*prefactor;
	  // = make_real2(saru.f(-1.0f, 1.0f),saru.f(-1.0f, 1.0f))*sqrt32;
	  noise.x = make_real2(saru.gf(0, complex_gaussian_sc));
	  // = make_real2(saru.f(-1.0f, 1.0f),saru.f(-1.0f, 1.0f))*sqrt32;
	  noise.y = make_real2(saru.gf(0, complex_gaussian_sc));
	  // = make_real2(saru.f(-1.0f, 1.0f),saru.f(-1.0f, 1.0f))*sqrt32;
	  noise.z = make_real2(saru.gf(0, complex_gaussian_sc));
	}
	/*Beware of nyquist points! They only appear with even cell dimensions
	  There are 8 nyquist points at most (cell=0,0,0 is excluded at the start of the kernel)
	  These are the 8 vertex of the inferior left cuadrant. The O points:
               +--------+--------+
              /|       /|       /|
             / |      / |      / |
            +--------+--------+  |
           /|  |    /|  |    /|  |
          / |  +---/-|--+---/-|--+
         +--------+--------+  |	/|
         |  |/ |  |  |/ |  |  |/ |
         |  O-----|--O-----|--+	 |
         | /|6 |  | /|7 |  | /|	 |
         |/ |  +--|/-|--+--|/-|--+
         O--------O--------+  |	/
         |5 |/    |4 |/    |  |/
         |  O-----|--O-----|--+
     ^   | / 3    | / 2    | /  ^
     |   |/       |/       |/  /
     kz  O--------O--------+  ky
         kx ->     1
	*/
	/*Handle nyquist points*/

	bool nyquist;
	{ //Is the current wave number a nyquist point?
	  bool isXnyquist = (cell.x == grid.cellDim.x - cell.x) && (grid.cellDim.x%2 == 0);
	  bool isYnyquist = (cell.y == grid.cellDim.y - cell.y) && (grid.cellDim.y%2 == 0);
	  bool isZnyquist = (cell.z == grid.cellDim.z - cell.z) && (grid.cellDim.z%2 == 0);

	  nyquist =  (isXnyquist && cell.y==0   && cell.z==0)  or  //1
               	     (isXnyquist && isYnyquist  && cell.z==0)  or  //2
               	     (cell.x==0  && isYnyquist  && cell.z==0)  or  //3
               	     (isXnyquist && cell.y==0   && isZnyquist) or  //4
               	     (cell.x==0  && cell.y==0   && isZnyquist) or  //5
               	     (cell.x==0  && isYnyquist  && isZnyquist) or  //6
               	     (isXnyquist && isYnyquist  && isZnyquist);    //7
	}

	if(nyquist){
	  /*Nyquist points are their own conjugates, so they must be real.
	    ||r||^2 = <x^2> = ||Real{z}||^2 = <Real{z}^2>·sqrt(2) =  prefactor*/
	  constexpr real nqsc = real(1.41421356237310); //sqrt(2)
	  noise.x.x *= nqsc; noise.x.y = 0;
	  noise.y.x *= nqsc; noise.y.y = 0;
	  noise.z.x *= nqsc; noise.z.y = 0;
	}
	/*Z = sqrt(B)·(I-k^k)·dW*/
	{// Compute for v_k wave number
	  const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);

	  const real invk2 = real(1.0)/dot(k,k);
	  /*Get my scaling factor B, Fourier representation of FCM*/
	  const real B = invk2/vis;
	  const real Bsq = sqrt(B/real(ncells));

	  cufftComplex3 factor = noise;
	  factor.x *= Bsq;
	  factor.y *= Bsq;
	  factor.z *= Bsq;

	  gridVelsFourier[icell] += projectFourier(k, factor);
	}
	/*Compute for conjugate v_{N-k} if needed*/

	/*Take care of conjugate wave number -> v_{Nx-kx,Ny-ky, Nz-kz}*/
	/*The special cases k_i=0 do not have conjugates, a.i N-k = N which is not stored*/

	if(nyquist) return; //Nyquist points do not have conjugates

	/*Conjugates are stored only when kx == Nx/2 or kx=0*/
	if(cell.x == grid.cellDim.x-cell.x or cell.x == 0){
	  /*The only case with x conjugates is when kx = Nx-kx or kx=0, so this line is not needed*/
	  //if(cell.x > 0) cell.x = grid.cellDim.x-cell.x;
	  /*k_i=N_i is not stored, so do not conjugate them, the necessary exclusions are at the start of the kernel*/
	  if(cell.y > 0) cell.y = grid.cellDim.y-cell.y;
	  if(cell.z > 0) cell.z = grid.cellDim.z-cell.z;

	  icell = grid.getCellIndex(cell);

	  const real3 k = cellToWaveNumber(cell, grid.cellDim, grid.box.boxSize);

	  const real invk2 = real(1.0)/dot(k,k);
	  /*Get my scaling factor B,  Fourier representation of FCM*/
	  const real B = invk2/vis;

	  const real Bsq = sqrt(B/real(ncells));
	  cufftComplex3 factor = noise;
	  /*v_{N-k} = v*_k, so the complex noise must be conjugated*/
	  factor.x.y *= real(-1.0);
	  factor.y.y *= real(-1.0);
	  factor.z.y *= real(-1.0);

	  factor.x *= Bsq;
	  factor.y *= Bsq;
	  factor.z *= Bsq;

	  gridVelsFourier[icell] += projectFourier(k, factor);
	}
      }


      struct toReal3{
	template<class vtype>
	inline __device__ real3 operator()(vtype q){ return make_real3(q);}
      };


    }

    //Spreads the particle quantity v to the grid, AKA applies the operator S to i.e the force.
    template<typename vtype>
    void FCM::spreadParticles(vtype *quantity, hipStream_t st){
      int numberParticles = pg->getNumberParticles();
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      real3* d_gridVels = (real3*)thrust::raw_pointer_cast(gridVelsFourier.data());

      auto tr = thrust::make_transform_iterator(quantity, FCM_ns::toReal3());

      ibm->spread(pos.begin(), tr, d_gridVels, numberParticles, st);
    }

    //Takes grid forcing and transforms it to grid velocity, also adds the fluctuations.
    //For that it solves the eq. \vec{v} = B·\vec{g}+fluct. -> \hat{\vec{v}} =  1/(\eta h^3) 1/k^2 \hat{P} · \hat{\vec{g}} + \hat{fluct}
    //Being \hat{P} = (I-\vec{k}\otimes \vec{k}) the projection onto the space of divergence free velocity fields.
    //v = FFTi·(B·FFTf·g + 1/√(h^3)·√B·dWw)
    //g is the fluid forcing which includes S·F and any other external force density on the fluid.
    void FCM::convolveFourier(hipStream_t st){
      CufftSafeCall(hipfftSetStream(cufft_plan_forward, st));
      CufftSafeCall(hipfftSetStream(cufft_plan_inverse, st));

      auto d_gridVels = thrust::raw_pointer_cast(gridVelsFourier.data());
      auto d_gridVelsFourier = thrust::raw_pointer_cast(gridVelsFourier.data());

      sys->log<System::DEBUG2>("[BDHI::FCM] Taking grid to wave space");
      {
	/*Take the grid spreaded forces and apply take it to wave space -> FFTf·S·F*/
	auto cufftStatus =
	  hipfftExecR2C(cufft_plan_forward,
		       (hipfftReal*)d_gridVels,
		       (hipfftComplex*)d_gridVelsFourier);
	if(cufftStatus != HIPFFT_SUCCESS){
	  sys->log<System::CRITICAL>("[BDHI::FCM] Error in forward CUFFT");
	}
      }
      sys->log<System::DEBUG2>("[BDHI::FCM] Wave space convolution");
      {
	/*Scale the wave space grid forces, transforming in velocities -> B·FFT·S·F*/
	//Launch a 3D grid of threads, a thread per cell.
	//Only the second half of the cells in the innermost (x) coordinate need to be processed, the rest are redundant and not used by cufft.

	dim3 NthreadsCells = dim3(8,8,8);
	dim3 NblocksCells;
	{
	  int ncellsx = grid.cellDim.x/2+1;
	  NblocksCells.x= (ncellsx/NthreadsCells.x + ((ncellsx%NthreadsCells.x)?1:0));
	  NblocksCells.y= grid.cellDim.y/NthreadsCells.y + ((grid.cellDim.y%NthreadsCells.y)?1:0);
	  NblocksCells.z= grid.cellDim.z/NthreadsCells.z + ((grid.cellDim.z%NthreadsCells.z)?1:0);
	}

	FCM_ns::forceFourier2Vel<<<NblocksCells, NthreadsCells, 0, st>>>
	  ((cufftComplex3*) d_gridVelsFourier, //Input: FFT·S·F
	   (cufftComplex3*) d_gridVelsFourier, //Output: B·FFT·S·F
	   viscosity,
	   grid);
	/*Add the stochastic noise to the fourier velocities if T>0 -> 1/√σ·√B·dWw */
	if(temperature > real(0.0)){
	  sys->log<System::DEBUG2>("[BDHI::FCM] Wave space brownian noise");
	  static ullint counter = 0; //Seed the rng differently each call
	  counter++;
	  sys->log<System::DEBUG2>("[BDHI::FCM] Wave space brownian noise");
	  real dV = grid.cellSize.x*grid.cellSize.y*grid.cellSize.z;
	  real prefactor = sqrt(2*temperature/(dt*dV)); //See eq. 53 in [1]
	  FCM_ns::fourierBrownianNoise<<<NblocksCells, NthreadsCells, 0, st>>>(
			//In: B·FFT·S·F -> Out: B·FFT·S·F + 1/√σ·√B·dWw
			(cufftComplex3*)d_gridVelsFourier,
			grid,
			prefactor, // 1/√σ· sqrt(2*T/dt),
			viscosity,
			seed, //Saru needs two seeds apart from thread id
			counter);
	}
      }
      sys->log<System::DEBUG2>("[BDHI::FCM] Going back to real space");
      {
	/*Take the fourier velocities back to real space ->  FFTi·(B·FFT·S·F + 1/√σ·√B·dWw )*/
	auto cufftStatus =
	  hipfftExecC2R(cufft_plan_inverse,
		       (hipfftComplex*)d_gridVelsFourier,
		       (hipfftReal*)d_gridVels);
	if(cufftStatus != HIPFFT_SUCCESS){
	  sys->log<System::CRITICAL>("[BDHI::FCM] Error in inverse CUFFT");
	}
      }
      CudaCheckError();

    }

    //Interpolates the velocities of the grid to the particle positions and adds it to an array
    //\vec{Mv} = h^3·St·\vec{v}
    void FCM::interpolateParticles(real3 *Mv, hipStream_t st){
      sys->log<System::DEBUG2>("[BDHI::FCM] Grid to particles");
      int numberParticles = pg->getNumberParticles();
      auto pos = pd->getPos(access::location::gpu, access::mode::read);
      real3* d_gridVels = (real3*)thrust::raw_pointer_cast(gridVelsFourier.data());

      ibm->gather(pos.begin(), Mv, d_gridVels, numberParticles, st);
    }

    /*Compute M·F and B·dW in Fourier space
      σ = dx*dy*dz; h^3 in [1]
      Mw·F + sqrt(Mw)·dWw = σ·St·FFTi·B·FFTf·S·F+ √σ·St·FFTi·√B·dWw =
      = σ·St·FFTi( B·FFTf·S·F + 1/√σ·√B·dWw)
    */
    template<typename vtype>
    void FCM::Mdot_far(real3 *Mv, vtype *v, hipStream_t st){
      sys->log<System::DEBUG1>("[BDHI::FCM] Computing MF wave space....");
      /*Clean gridVels*/
      {
	int ncells = grid.cellDim.x*grid.cellDim.y*grid.cellDim.z;
	int BLOCKSIZE = 128;
	int Nthreads = BLOCKSIZE<ncells?BLOCKSIZE:ncells;
	int Nblocks  =  ncells/Nthreads +  ((ncells%Nthreads!=0)?1:0);

	sys->log<System::DEBUG2>("[BDHI::FCM] Setting vels to zero...");
	//Note that the same storage space is used for Fourier and real space
	//The real space is the only one that needs to be cleared.
	auto d_gridVels = (real3*)thrust::raw_pointer_cast(gridVelsFourier.data());
	fillWithGPU<<<Nblocks, Nthreads, 0, st>>>(d_gridVels,
						  make_real3(0), ncells);
      }

      spreadParticles(v, st);
      convolveFourier(st);
      interpolateParticles(Mv, st);
      // real3 mv;
      // hipMemcpy(&mv, Mv, sizeof(real3), hipMemcpyDeviceToHost);
      // sys->log<System::CRITICAL>("[BDHI::FCM] %g %g %g", mv.x, mv.y, mv.z);
    }

    void FCM::computeMF(real3* MF, hipStream_t st){
      sys->log<System::DEBUG1>("[BDHI::FCM] Computing MF....");
      auto force = pd->getForce(access::location::gpu, access::mode::read);
      Mdot<real4>(MF, force.raw(), st);
    }

    void FCM::computeBdW(real3* BdW, hipStream_t st){
      //This part is included in Fourier space when computing MF
    }


  }
}
